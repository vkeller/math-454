/* Hello Cuda example */
/* Intro to GPU tutorial */
/* SCV group */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 4
#define BLOCK_WIDTH 8


/* Function executed on device (GPU */
__global__ void hello( void) {
  printf("\tHello from GPU: thread %d and block %d\n", threadIdx.x, blockIdx.x);

} 

/* Main function, executed on host (CPU) */
int main( void) {

  /* print message from CPU */
  printf( "Hello Cuda!\n" );

  /* execute function on device */
  hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

  /* wait until all threads finish their job */
  hipDeviceSynchronize();

  /* print message from CPU */
  printf( "Welcome back to CPU!\n" );

  return (0);
}
