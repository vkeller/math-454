#include "hipsparse.h"

/******************/
/* CUSPARSETCSRMV */
/******************/
hipsparseStatus_t cusparseTcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const float *alpha,
	const hipsparseMatDescr_t descrA, const float *csrValA, const int *csrRowPtrA, const int *csrColIndA, const float *x, const float *beta, float *y)  {
	return hipsparseScsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, beta, y);
}

hipsparseStatus_t cusparseTcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const double *alpha,
	const hipsparseMatDescr_t descrA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, const double *x, const double *beta, 
	double *y) {
	return hipsparseDcsrmv(handle, transA, m, n, nnz, alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, beta, y); }
