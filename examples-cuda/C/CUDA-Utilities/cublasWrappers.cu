
#include <hip/hip_runtime.h>
#include "hipblas.h"

/**************/
/* CUBLASTDOT */
/**************/
hipblasStatus_t cublasTdot(hipblasHandle_t handle, int n, const float *x, int incx, const float *y, int incy, float *result) {
	return hipblasSdot(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublasTdot(hipblasHandle_t handle, int n, const double *x, int incx, const double *y, int incy, double *result) {
	return hipblasDdot(handle, n, x, incx, y, incy, result);
}

/***************/
/* CUBLASTAXPY */
/***************/
hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const float *alpha, const float *x, int incx, float *y, int incy) {
	return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y, int incy) {
	return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

